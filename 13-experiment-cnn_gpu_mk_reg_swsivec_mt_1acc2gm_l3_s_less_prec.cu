
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
//#include <cooperative_groups.h>
#include <math.h>
#include <string.h>
#include <sstream>
#include <fstream>
//#include <bits/stdc++.h>
//#include <stdlib.h>
//#include <time.h>
using namespace std;
//using namespace cooperative_groups;

/***DEFINING THE DEFINES FOR THE ARRAY INDICES****************************/
//#define N 32
#define C 256
#define H 15
#define W 15
#define R 3
#define S 3
#define M 384
#define E 13
#define F 13
#define U 1


__global__ void red_ch(float* d_r, float* d_o, int num_ch, int num_img, int num_wt)
{
//printf("gpu2 started\n");
float red_sum = 0;
int row = threadIdx.y; int col = threadIdx.x;
for(int i=0; i<num_ch; i++)
{       
        red_sum += d_o[i*(num_wt*num_img*blockDim.x*blockDim.y)+blockIdx.x*num_wt*blockDim.x*blockDim.y+blockIdx.y*blockDim.x*blockDim.y+row*blockDim.x+col] ;
}
d_r[blockIdx.x*num_wt*blockDim.x*blockDim.y+blockIdx.y*blockDim.x*blockDim.y+row*blockDim.x+col] = red_sum;
}
__global__
void ew_gpu_mmul(float* d_o, float* d_i, float* d_w, int width, int height, int stride, int ip_height, int wt_width, int num_wt,int num_img, int num_ch)
{//printf("gpu started\n");
__shared__ float s_w[R*S];
__shared__ float s_i[H*W];
int row = threadIdx.y; int col = threadIdx.x;
if(row*width+col<R*S)
{
s_w[row*width+col] = d_w[blockIdx.y*num_ch*wt_width*wt_width+blockIdx.z*wt_width*wt_width+(row*width+col)];
}
{
int s_i_idx = row*blockDim.x+col;
s_i[s_i_idx] = d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx];
//s_i[s_i_idx+169] = d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx+169];
//s_i[s_i_idx+338] = d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx+338];
//s_i[s_i_idx+507] = d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx+507];
if(s_i_idx+169 < H*W)
s_i[s_i_idx+169] = d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx+169];
}
__syncthreads();

float prod = 0;
if((row<height) && (col<width))//earlier it was num_wt*height & num_img*width
{
for (int i=0; i<wt_width; i++){
	float3 ip = *((float3*)(s_i+(stride*row+i)*ip_height+stride*col));
        float3 wt = *((float3*)(s_w+i*wt_width));
	prod += ip.x*wt.x+ip.y*wt.y+ip.z*wt.z;
        __syncthreads();
}
if(prod>=0)
d_o[0*(num_wt*num_img*blockDim.x*blockDim.y)+blockIdx.x*num_wt*blockDim.x*blockDim.y+blockIdx.y*blockDim.x*blockDim.y+row*blockDim.x+col] += prod;
if(row*width+col<R*S){
     s_w[(row*width+col)] = 0;
__syncthreads();
}
}
}

void element_wise_mmul(float* output, float* input, float* weight, int batch_size)
{
int x,y,i,j,m,n,k;
for(n=0; n<batch_size; n++){
for (m=0 ; m<M; m++){
 for (x=0; x<F; x++){
         for(y=0; y<E; y++){
            //    OP[x][y] = 0; // adding bias to output
                 for (i=0; i<R; i++){
                         for (j=0; j<S; j++){
				for(k=0; k<C; k++){
				float ip = input[n*C*H*W+k*H*W+(U*x+i)*H+(U*y+j)];
				float wt = weight[m*C*R*S+k*R*S+i*S+j];

				float prod = ip*wt;
				if(prod>=0)
				output[n*E*F*M+m*E*F+x*E+y] += prod;
                                 //OP[x][y] += IP[U*x+i][U*y+j]*WT[i][j];
                                                         }}
                                                 }
                                        }
                                         }
 

}
}
}
int main(int argc, char* argv[])
{
int batch_size = atoi(argv[1]);
/*************INITALIZING MATRICES*********************************/
float *IP = (float*) malloc(batch_size*C*H*W*sizeof(float));
//float IP[H][W];
float *OP = (float*) malloc(batch_size*M*F*E*sizeof(float));
//float OP[F][E];
float *OPG = (float*) malloc(batch_size*M*F*E*sizeof(float));
float *WT = (float*) malloc(M*C*R*S*sizeof(float));
//float WT[R][S];
float* d_o;
float* d_i;
float* d_w;
float* d_r;
//clock_t cpu_start, gpu_start, cpu_end, gpu_end;
//int a,b,c,d;
int c,d,m,n,k;
/*INITIALIZING WEIGHT MATRIX*/
for (m=0; m<M; m++){
for(k=0;k<C;k++){
for (c=0; c<R; c++){
	for(d=0; d<S; d++){
		//WT[c][d] = 2.0;
		//WT[m*C*R*S+k*R*S+c*S+d] = (int)k+1;
		WT[m*C*R*S+k*R*S+c*S+d] = (float)rand()/(float)(RAND_MAX+1.0);
}	
}
}
}
/*INITIALIZING OUTPUT MATRIX*/
for (n=0; n<batch_size;n++){
for (m=0; m<M; m++){
for (c=0; c<F; c++){
	for(d=0; d<E; d++){
		//OP[c][d] = 0;
		OP[n*M*F*E+m*F*E+c*E+d] = 0;
}
}	
}
}
/*INITIALIZING INPUT MATRIX*/
for (n=0; n<batch_size; n++){
for(k=0;k<C;k++){
for (c=0; c<H; c++){
	for(d=0; d<W; d++){
	//	IP[c][d] = (a+b+c+d);
	if ((c==0) || (d==0) || (c==14) || (d==14))
        IP[n*C*H*W+k*H*W+c*W+d] = 0;
        else
        IP[n*C*H*W+k*H*W+c*W+d] = (float)rand()/(RAND_MAX+1.0);
	
}
}	
}
}
if(hipSuccess != hipMalloc((void**) &d_i,batch_size*C*H*W*sizeof(float)))
{
printf("error in d_i malloc\n");
}
hipMemcpy(d_i, IP, batch_size*C*H*W*sizeof(float), hipMemcpyHostToDevice);
if(hipSuccess != hipMalloc((void**) &d_w, M*C*R*S*sizeof(float)))
{
printf("error in d_w malloc\n");	
}
hipMemcpy(d_w, WT, M*C*R*S*sizeof(float), hipMemcpyHostToDevice);
if(hipSuccess != hipMalloc((void**) &d_o,(long int)batch_size*M*E*F*sizeof(float)))
{
printf("error in d_o malloc\n");
}
if(hipSuccess != hipMalloc((void**) &d_r,batch_size*M*E*F*sizeof(float)))
{
printf("error in d_r malloc\n");
}

//cpu_start = clock();
//element_wise_mmul(OP, IP, WT, batch_size);
printf("cpu done\n");
//cpu_end = clock();
dim3 dimGrid(batch_size,384,256);
dim3 dimBlock(13,13,1);
//dim3 dimGridRed(batch_size,384,1);
//dim3 dimBlockRed(13,13,1);
//int op_height = 3; int op_width = 3; int stride = 1; int ip_height = 4;int wt_height = 2; int num_wt = 96; int num_img = 1; int num_ch = 384;
//gpu_start = clock();
ew_gpu_mmul<<<dimGrid, dimBlock>>>(d_o,d_i,d_w,13,13,1,15,3,384,batch_size,256);
hipDeviceSynchronize();
//red_ch<<<dimGridRed, dimBlockRed>>>(d_r,d_o,256,batch_size,384);
//gpu_end = clock();
//void *kernelArgs[] = {(void *)&d_o, (void *)&d_i, (void *)&d_w,(void *)&op_height, (void *)&op_width, (void *)&stride, (void *)&ip_height,(void *)&wt_height, (void *)&num_wt, (void *)&num_img, (void *)&num_ch };
//cudaLaunchCooperativeKernel((void*)ew_gpu_mmul,dimGrid,dimBlock,kernelArgs,0,NULL);
//cudaDeviceSynchronize();
hipMemcpy(OPG,d_o,batch_size*M*E*F*sizeof(float), hipMemcpyDeviceToHost);

/**print outputs**/
//int e,f,g,h;
int g,h,s,u;
float max_error = 0;
string filename = "layer_3_"+to_string(batch_size);
ifstream fin(filename.c_str());
string line ;
//for (t=0;t<C;t++){
for (u=0;u<batch_size;u++){
for (s=0;s<M;s++){
for (g=0; g<F; g++){
	for(h=0; h<E; h++){	
	getline(fin,line);
        float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-atof(line.c_str()));
//	float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h]);
	if(error > max_error)
	max_error = error;
 //     printf("the output is %f for index %d, %d,%d,%d.\n",OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
   //    printf("diff CPU and GPU is %f for index %d,%d,%d,%d.\n", OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
     //   printf("the output from GPU  is %f for index,%d,%d,%d,%d.\n",OPG[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
}
}
}
}
fin.close();
printf("max error is %f\n", max_error);
//}
//cout<<"time taken by cpu call is "<<((double)(cpu_end-cpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;
//cout<<"time taken by gpu call is "<<((double)(gpu_end-gpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;

hipFree(d_o);
hipFree(d_i);
hipFree(d_w);
hipFree(d_r);
free(OPG);
free(IP);
free(WT);
free(OP);	
return 0;
}

