
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
//#include <cooperative_groups.h>
#include <math.h>
#include <string.h>
#include <sstream>
#include <fstream>
//#include <bits/stdc++.h>
//#include <stdlib.h>
//#include <time.h>
using namespace std;
//using namespace cooperative_groups;

/***DEFINING THE DEFINES FOR THE ARRAY INDICES****************************/
//#define N 32
#define C 96
#define H 31
#define W 31
#define R 5
#define S 5
#define M 256
#define E 27
#define F 27
#define U 1


__global__ void red_ch(float* d_r, float* d_o, int num_ch, int num_img, int num_wt)
{
//printf("gpu2 started\n");
float red_sum = 0;
int row = threadIdx.y; int col = threadIdx.x;
for(int i=0; i<num_ch; i++)
{       
        red_sum += d_o[i*(num_wt*num_img*blockDim.x*blockDim.y)+blockIdx.x*num_wt*blockDim.x*blockDim.y+blockIdx.y*blockDim.x*blockDim.y+row*blockDim.x+col] ;
}
d_r[blockIdx.x*num_wt*blockDim.x*blockDim.y+blockIdx.y*blockDim.x*blockDim.y+row*blockDim.x+col] = red_sum;
}
__global__
void ew_gpu_mmul(float* d_o, float* d_i, float* d_w, int width, int height, int stride, int ip_height, int wt_width, int num_wt,int num_img, int num_ch)
{//printf("gpu started\n");
__shared__ float s_w[R*S];
__shared__ float s_i[H*W];
int row = threadIdx.y; int col = threadIdx.x;
if(row*width+col<R*S)
{
s_w[row*width+col] = d_w[blockIdx.y*num_ch*wt_width*wt_width+blockIdx.z*wt_width*wt_width+(row*width+col)];
}
{
int s_i_idx = row*blockDim.x+col;
s_i[s_i_idx] = d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx];
if(s_i_idx+729 < H*W)
s_i[s_i_idx+729]= d_i[blockIdx.x*num_ch*ip_height*ip_height+blockIdx.z*ip_height*ip_height+s_i_idx+729];
}
__syncthreads();
float prod = 0;
if((row<height) && (col<width))//earlier it was num_wt*height & num_img*width
{
for (int i=0; i<wt_width; i++){
	float3 ip1 = *((float3*)(s_i+(stride*row+i)*ip_height+stride*col)); float3 wt1 = *((float3*)(s_w+i*wt_width));
	float3 ip2 = *((float3*)(s_i+(stride*row+i)*ip_height+stride*col+3));float3 wt2 = *((float3*)(s_w+i*wt_width+3));
	prod += ip1.x*wt1.x+ip1.y*wt1.y+ip1.z*wt1.z+ip2.x*wt2.x+ip2.y*wt2.y;
        __syncthreads();
}
if(prod>=0)
d_o[blockIdx.z*(num_wt*num_img*blockDim.x*blockDim.y)+blockIdx.x*num_wt*blockDim.x*blockDim.y+blockIdx.y*blockDim.x*blockDim.y+row*blockDim.x+col] = prod;
if(row*width+col<R*S){
     s_w[(row*width+col)] = 0;
__syncthreads();
}
}
}

void element_wise_mmul(float* output, float* input, float* weight, int batch_size)
{
int x,y,i,j,m,n,k;
for(n=0; n<batch_size; n++){
for (m=0 ; m<M; m++){
 for (x=0; x<F; x++){
         for(y=0; y<E; y++){
            //    OP[x][y] = 0; // adding bias to output
                 for (i=0; i<R; i++){
                         for (j=0; j<S; j++){
				for(k=0; k<C; k++){
				float ip = input[n*C*H*W+k*H*W+(U*x+i)*H+(U*y+j)];
				float wt = weight[m*C*R*S+k*R*S+i*S+j];

				float prod = ip*wt;
				if(prod>=0)
				output[n*E*F*M+m*E*F+x*E+y] += prod;
                                 //OP[x][y] += IP[U*x+i][U*y+j]*WT[i][j];
                                                         }}
                                                 }
                                        }
                                         }
 

}
}
}
int main(int argc, char* argv[])
{
int batch_size = atoi(argv[1]);
/*************INITALIZING MATRICES*********************************/
float *IP = (float*) malloc(batch_size*C*H*W*sizeof(float));
//float IP[H][W];
float *OP = (float*) malloc(batch_size*M*F*E*sizeof(float));
//float OP[F][E];
float *OPG = (float*) malloc(batch_size*M*F*E*sizeof(float));
float *WT = (float*) malloc(M*C*R*S*sizeof(float));
//float WT[R][S];
float* d_o;
float* d_i;
float* d_w;
float* d_r;
//clock_t cpu_start, gpu_start, cpu_end, gpu_end;
//int a,b,c,d;
int c,d,m,n,k;
/*INITIALIZING WEIGHT MATRIX*/
for (m=0; m<M; m++){
for(k=0;k<C;k++){
for (c=0; c<R; c++){
	for(d=0; d<S; d++){
		//WT[c][d] = 2.0;
		//WT[m*C*R*S+k*R*S+c*S+d] = (int)k+1;
		WT[m*C*R*S+k*R*S+c*S+d] = (float)rand()/(float)(RAND_MAX+1.0);
}	
}
}
}
/*INITIALIZING OUTPUT MATRIX*/
for (n=0; n<batch_size;n++){
for (m=0; m<M; m++){
for (c=0; c<F; c++){
	for(d=0; d<E; d++){
		//OP[c][d] = 0;
		OP[n*M*F*E+m*F*E+c*E+d] = 0;
}
}	
}
}
/*INITIALIZING INPUT MATRIX*/
for (n=0; n<batch_size; n++){
for(k=0;k<C;k++){
for (c=0; c<H; c++){
	for(d=0; d<W; d++){
	if ((c<=1) || (d<=1) || (c>=29) || (d>=29))
        IP[n*C*H*W+k*H*W+c*W+d] = 0;
        else
        IP[n*C*H*W+k*H*W+c*W+d] = (float)rand()/(RAND_MAX+1.0);

	//	IP[c][d] = (a+b+c+d);
		//IP[n*C*H*W+k*H*W+c*W+d] = (float)(c+d)/255;
}
}	
}
}
if(hipSuccess != hipMalloc((void**) &d_i,batch_size*C*H*W*sizeof(float)))
{
printf("error in d_i malloc\n");
}
hipMemcpy(d_i, IP, batch_size*C*H*W*sizeof(float), hipMemcpyHostToDevice);
if(hipSuccess != hipMalloc((void**) &d_w, M*C*R*S*sizeof(float)))
{
printf("error in d_w malloc\n");	
}
hipMemcpy(d_w, WT, M*C*R*S*sizeof(float), hipMemcpyHostToDevice);
if(hipSuccess != hipMalloc((void**) &d_o,(long int)C*batch_size*M*E*F*sizeof(float)))
{
printf("error in d_o malloc\n");
}
if(hipSuccess != hipMalloc((void**) &d_r,batch_size*M*E*F*sizeof(float)))
{
printf("error in d_r malloc\n");
}

//cpu_start = clock();
//element_wise_mmul(OP, IP, WT,batch_size);
printf("cpu done\n");
//cpu_end = clock();
dim3 dimGrid(batch_size,256,96);
dim3 dimBlock(27,27,1);
dim3 dimGridRed(batch_size,256,1);
dim3 dimBlockRed(27,27,1);
//int op_height = 3; int op_width = 3; int stride = 1; int ip_height = 4;int wt_height = 2; int num_wt = 96; int num_img = 1; int num_ch = 384;
//gpu_start = clock();
//cudaFuncSetSharedMemConfig(ew_gpu_mmul,cudaSharedMemBankSizeEightByte);
ew_gpu_mmul<<<dimGrid, dimBlock>>>(d_o,d_i,d_w,27,27,1,31,5,256,batch_size,96);
hipDeviceSynchronize();
red_ch<<<dimGridRed, dimBlockRed>>>(d_r,d_o,96,batch_size,256);
//gpu_end = clock();
//void *kernelArgs[] = {(void *)&d_o, (void *)&d_i, (void *)&d_w,(void *)&op_height, (void *)&op_width, (void *)&stride, (void *)&ip_height,(void *)&wt_height, (void *)&num_wt, (void *)&num_img, (void *)&num_ch };
//cudaLaunchCooperativeKernel((void*)ew_gpu_mmul,dimGrid,dimBlock,kernelArgs,0,NULL);
//cudaDeviceSynchronize();
hipMemcpy(OPG,d_r,batch_size*M*E*F*sizeof(float), hipMemcpyDeviceToHost);

/**print outputs**/
//int e,f,g,h;
int g,h,s,u;
float max_error = 0;
string filename = "layer_2_"+to_string(batch_size);
ifstream fin(filename.c_str());
string line ;
//for (t=0;t<C;t++){
for (u=0;u<batch_size;u++){
for (s=0;s<M;s++){
for (g=0; g<F; g++){
	for(h=0; h<E; h++){	
	getline(fin,line);
        float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-atof(line.c_str()));
	//float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h]);
	if(error > max_error)
	max_error = error;
 //     printf("the output is %f for index %d, %d,%d,%d.\n",OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
   //    printf("diff CPU and GPU is %f for index %d,%d,%d,%d.\n", OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
     //   printf("the output from GPU  is %f for index,%d,%d,%d,%d.\n",OPG[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
}
}
}
}
fin.close();
printf("max error is %f\n", max_error);
//}
//cout<<"time taken by cpu call is "<<((double)(cpu_end-cpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;
//cout<<"time taken by gpu call is "<<((double)(gpu_end-gpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;

hipFree(d_o);
hipFree(d_i);
hipFree(d_w);
hipFree(d_r);
free(OPG);
free(IP);
free(WT);
free(OP);	
return 0;
}

