
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <string.h>
#include <sstream>
#include <fstream>
//#include <bits/stdc++.h>
//#include <stdlib.h>
//#include <time.h>
using namespace std;
/***DEFINING THE DEFINES FOR THE ARRAY INDICES****************************/
//#define N 1
#define C 256
#define H 15
#define W 15
#define R 3
#define S 3
#define M 384
#define E 13
#define F 13
#define U 1
__global__ 
void ew_gpu_mmul(float* d_o, float* d_i, float* d_w, int width, int height, int stride, int ip_height, int wt_width, int num_wt,int num_img, int num_ch)
{
int row = threadIdx.y; int col = threadIdx.x;
{
if((row<height) && (col<width))//earlier it was num_wt*height & num_img*width
{
float prod = 0;
for (int i=0; i<wt_width; i++){
  for (int j=0; j<wt_width; j++){
   for(int k=0; k<num_ch; k++){
    float ip = d_i[blockIdx.x*num_ch*ip_height*ip_height+k*ip_height*ip_height+(stride*(row)+i)*ip_height+(stride*(col)+j)];
       float wt = d_w[(blockIdx.y)*num_ch*wt_width*wt_width+k*wt_width*wt_width+(i*wt_width+j)];
//	float wt = s_w[k*wt_width*wt_width+(i*wt_width+j)];	
	prod += ip*wt;
}
}
}
if(prod>=0)
d_o[blockIdx.x*num_wt*blockDim.x*blockDim.y+(blockIdx.y)*blockDim.x*blockDim.y+row*blockDim.x+col] = prod;
}
}
}
void element_wise_mmul(float* output, float* input, float* weight, int batch_size)
{
int x,y,i,j,m,n,k;
for(n=0; n<batch_size; n++){
for (m=0 ; m<M; m++){
 for (x=0; x<F; x++){
         for(y=0; y<E; y++){
            //    OP[x][y] = 0; // adding bias to output
                 for (i=0; i<R; i++){
                         for (j=0; j<S; j++){
				for(k=0; k<C; k++){
				float ip = input[n*C*H*W+k*H*W+(U*x+i)*H+(U*y+j)];
				float wt = weight[m*C*R*S+k*R*S+i*S+j];

				float prod = ip*wt;
				if(prod>=0)
				output[n*E*F*M+m*E*F+x*E+y] += prod;
                                 //OP[x][y] += IP[U*x+i][U*y+j]*WT[i][j];
                                                         }}
                                                 }
                                        }
                                         }
 

}
}
}
int main(int argc, char* argv[])
{
int batch_size = atoi(argv[1]);
/*************INITALIZING MATRICES*********************************/
float *IP = (float*) malloc(batch_size*C*H*W*sizeof(float));
//float IP[H][W];
float *OP = (float*) malloc(batch_size*M*F*E*sizeof(float));
//float OP[F][E];
float *OPG = (float*) malloc(batch_size*M*F*E*sizeof(float));
float *WT = (float*) malloc(M*C*R*S*sizeof(float));
//float WT[R][S];
float* d_o;
float* d_i;
float* d_w;

//clock_t cpu_start, gpu_start, cpu_end, gpu_end;
//int a,b,c,d;
int c,d,m,n,k;
/*INITIALIZING WEIGHT MATRIX*/
for (m=0; m<M; m++){
for(k=0;k<C;k++){
for (c=0; c<R; c++){
	for(d=0; d<S; d++){
		//WT[c][d] = 2.0;
	WT[m*C*R*S+k*R*S+c*S+d] = (float)rand()/(RAND_MAX+1.0);
//		WT[m*C*R*S+k*R*S+c*S+d] = 1;

}
}	
}
}
/*INITIALIZING OUTPUT MATRIX*/
for (n=0; n<batch_size;n++){
for (m=0; m<M; m++){
for (c=0; c<F; c++){
	for(d=0; d<E; d++){
		//OP[c][d] = 0;
		OP[n*M*F*E+m*F*E+c*E+d] = 0;
}
}	
}
}
/*INITIALIZING INPUT MATRIX*/
for (n=0; n<batch_size; n++){
for(k=0;k<C;k++){
for (c=0; c<H; c++){
	for(d=0; d<W; d++){
	//	IP[c][d] = (a+b+c+d);
	if ((c==0) || (d==0) || (c==14) || (d==14))
        IP[n*C*H*W+k*H*W+c*W+d] = 0;
        else
        IP[n*C*H*W+k*H*W+c*W+d] = (float)rand()/(RAND_MAX+1.0);

}
}	
}
}
hipMalloc((void**) &d_i,batch_size*C*H*W*sizeof(float));
hipMemcpy(d_i, IP, batch_size*C*H*W*sizeof(float), hipMemcpyHostToDevice);
hipMalloc((void**) &d_w, M*C*R*S*sizeof(float));
hipMemcpy(d_w, WT, M*C*R*S*sizeof(float), hipMemcpyHostToDevice);
hipMalloc((void**) &d_o, batch_size*M*E*F*sizeof(float));
//cpu_start = clock();
//element_wise_mmul(OP, IP, WT,batch_size);
//cpu_end = clock();
dim3 dimGrid(batch_size,384,1);
dim3 dimBlock(13,13,1);
//gpu_start = clock();
ew_gpu_mmul<<<dimGrid, dimBlock>>>(d_o,d_i,d_w,13,13,1,15,3,384,batch_size,256);
//gpu_end = clock();
hipMemcpy(OPG,d_o, batch_size*M*E*F*sizeof(float), hipMemcpyDeviceToHost);
float max_error = 0;
string filename = "layer_3_"+to_string(batch_size);
ifstream fin(filename.c_str());
string line ;
int g,h,s,u;
for (u=0;u<batch_size;u++){
for (s=0;s<M;s++){
for (g=0; g<F; g++){
	for(h=0; h<E; h++){	
	getline(fin,line);
        float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-atof(line.c_str()));
	//float error = abs(OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h]);
	if(error > max_error)
	max_error = error; 
 //     printf("the output is %f for index %d, %d,%d,%d.\n",OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
   //    printf("diff CPU and GPU is %f for index %d,%d,%d,%d.\n", OPG[u*M*F*E+s*E*F+g*E+h]-OP[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
 //       printf("the output from GPU  is %f for index %d,%d,%d,%d.\n",OPG[u*M*F*E+s*E*F+g*E+h],u,s,g,h);
}
}
}
}
fin.close();
printf("max error is %f\n", max_error);
//cout<<"time taken by cpu call is "<<((double)(cpu_end-cpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;
//cout<<"time taken by gpu call is "<<((double)(gpu_end-gpu_start))/CLOCKS_PER_SEC<<"secs"<<endl;

hipFree(d_o);
hipFree(d_i);
hipFree(d_w);
free(OPG);
free(IP);
free(WT);
free(OP);	
return 0;
}

