
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <string.h>

/***DEFINING THE DEFINES FOR THE ARRAY INDICES****************************/
//#define N 1
#define C 384
#define H 15
#define W 15
#define R 3
#define S 3
#define M 256
#define E 13
#define F 13
#define U 1
using namespace std;
int main(int argc, char* argv[])
{
int batch_size = atoi(argv[1]);
/*************INITALIZING MATRICES*********************************/
float IP[batch_size][C][H][W];
float OP[batch_size][M][F][E];
float WT[M][C][R][S];
int a,b,c,d;
/*INITIALIZING WEIGHT MATRIX*/
for(a=0; a<M; a++){
	for (b=0; b<C; b++){
		for (c=0; c<R; c++){
			for(d=0; d<S; d++){
				WT[a][b][c][d] = (float)rand()/(float)(RAND_MAX+1.0);
}
}	
}
}
//printf("hello after weight\n");
/*INITIALIZING OUTPUT MATRIX*/
for(a=0; a<batch_size; a++){
	for (b=0; b<M; b++){
		for (c=0; c<F; c++){
			for(d=0; d<E; d++){
				OP[a][b][c][d] = 0;
}
}	
}
}
//printf("hello after op init\n");
/*INITIALIZING INPUT MATRIX*/
for(a=0; a<batch_size; a++){
	for (b=0; b<C; b++){
		for (c=0; c<H; c++){
			for(d=0; d<W; d++){
				if((c==0)||(d==0)||(c==14)||(d==14))
				IP[a][b][c][d] = 0;//zero padding to eliminate spurious values along the edges to align stride and filter boundary properly with image boundary
				else
				IP[a][b][c][d] = (float)rand()/(float)(RAND_MAX+1.0);;
}
}	
}
}
printf("hello after ininting input mat\n");
/***NAIVE 7 LAYER LOOP IMPLEMENTATION***/
int n,m,x,y,i,j,k;
for(n=0; n<batch_size; n++){
	for (m=0; m<M; m++){
		for (x=0; x<F; x++){
			for(y=0; y<E; y++){
				OP[n][m][x][y] = 0; // adding bias to output
				for (i=0; i<R; i++){
					for (j=0; j<S; j++){
						for(k=0; k<C; k++){
							OP[n][m][x][y] += IP[n][k][U*x+i][U*y+j]*WT[m][k][i][j];
							}
						}
					}
                 /*****ACTIVATION FUNCTION-RELU*******/
if(OP[n][m][x][y] < 0) 
	OP[n][m][x][y] = 0;

					}
				}	
			}
		}
/**print outputs**/
string filename ="layer_5_"+to_string(batch_size);
FILE *fp=fopen(filename.c_str(),"w+");

int e,f,g,h;
for(e=0; e<batch_size; e++){
	for (f=0; f<M; f++){
		for (g=0; g<F; g++){
			for(h=0; h<E; h++){
				//printf("the output is %f for index %d,%d,%d,%d.\n",OP[e][f][g][h], e,f,g,h);
				 //fprintf(fp,"%f,%d,%d,%d,%d.\n",OP[e][f][g][h], e,f,g,h);
				fprintf(fp,"%f\n",OP[e][f][g][h]);


}
}	
}
}
fclose(fp);
return 0;
}


